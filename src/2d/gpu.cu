#include "hip/hip_runtime.h"
#include "2d_utils.h"

#ifdef __INTELLISENSE__
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#endif

#include <mma.h>

#include <iostream>
#include <fstream>
#include <chrono>
#include "precision.h"
// #include "../utils.h"

#define DEBUG

using namespace nvcuda;

#define BLOCK_SIZE_ROW 32
#define BLOCK_SIZE_COL 128  // 64
#define HALO 3
#define D_BLOCK_SIZE_COL (BLOCK_SIZE_COL + HALO * 2)    // 128 + 6 = 134
#define D_BLOCK_SIZE_ROW (BLOCK_SIZE_ROW + HALO * 2)    // 32 + 6  = 38
#define PAD 2
#define SM_SIZE_COL (7 * D_BLOCK_SIZE_ROW + PAD)    // 7 * 38 + 2  = 266
#define SM_SIZE_ROW (D_BLOCK_SIZE_COL / 8)          // 134 / 8     = 16
#define UNIT_LENGTH 7
#define TENSOR_CORE_M 16 // 8
#define TENSOR_CORE_N 16 // 8
#define TENSOR_CORE_K 8 // 4
#define IDX(x, y, ldm) ((x) * (ldm) + (y))
#define WARP_PER_BLOCK 8
// #define ACCS_PER_WARP (BLOCK_SIZE_COL * BLOCK_SIZE_ROW / 64 / WARP_PER_BLOCK)
#define MMA_NUM 7 // 13
#define ceild(n,d)	(((n)-1)/(d) + 1)

__constant__ real_t param_matrix_d[2 * MMA_NUM * TENSOR_CORE_M * TENSOR_CORE_K];


__global__ void kernel2d_fp32 (const float * __restrict__ in, float * __restrict__ out, const int ldm, const int * __restrict__ lookup_table1, const int * __restrict__ lookup_table2) {
    
    __shared__ __align__(32) float sharedmem[2][SM_SIZE_ROW * SM_SIZE_COL];
    int begin = IDX(blockIdx.x * BLOCK_SIZE_ROW, blockIdx.y * BLOCK_SIZE_COL + 1, ldm);
    int tid = threadIdx.x;
    int totalThreads = blockDim.x;
    int warp_id = threadIdx.x / 32;

    // Load data into shared memory using lookup tables
    /*
        Data is loaded from global memory, in which resides the original input array.
        When loading into shared memory, we use lookup tables to apply the s2r layout.
        Data in shared memory has the stencil2row layout.
    */
#pragma unroll
    for (int i = tid; i < D_BLOCK_SIZE_ROW * D_BLOCK_SIZE_COL; i += totalThreads) {
        int row = i / D_BLOCK_SIZE_COL;
        int col = i % D_BLOCK_SIZE_COL;
        sharedmem[0][lookup_table1[i]] = in[begin + IDX(row, col, ldm)];
        sharedmem[1][lookup_table2[i]] = in[begin + IDX(row, col, ldm)];
    }
    __syncthreads();


    wmma::fragment<wmma::matrix_b, 16, 16, 8, wmma::precision::tf32, wmma::row_major> param_frag[2][MMA_NUM];
#pragma unroll
    for (int i = 0; i < MMA_NUM; i++) {
        wmma::load_matrix_sync(param_frag[0][i], param_matrix_d + i * TENSOR_CORE_M * TENSOR_CORE_K, TENSOR_CORE_M);
        wmma::load_matrix_sync(param_frag[1][i], param_matrix_d + (MMA_NUM + i) * TENSOR_CORE_M * TENSOR_CORE_K, TENSOR_CORE_M);
    }

    wmma::fragment<wmma::accumulator, 16, 16, 8, float> acc_frag;
    wmma::fragment<wmma::matrix_a, 16, 16, 8, wmma::precision::tf32, wmma::row_major> in_frag;
    for (int col = warp_id * 28; col < warp_id * 28 + 28; col += UNIT_LENGTH) {
        wmma::fill_fragment(acc_frag, 0.0);
#pragma unroll
        for (int compute_idx = 0; compute_idx < MMA_NUM; compute_idx++) {
            wmma::load_matrix_sync(in_frag, sharedmem[0] + (compute_idx * TENSOR_CORE_K + col), SM_SIZE_COL);
            wmma::mma_sync(acc_frag, in_frag, param_frag[0][compute_idx], acc_frag);
        }
#pragma unroll
        for (int compute_idx = 0; compute_idx < MMA_NUM; compute_idx++) {
            wmma::load_matrix_sync(in_frag, sharedmem[1] + (compute_idx * TENSOR_CORE_K + col), SM_SIZE_COL);
            wmma::mma_sync(acc_frag, in_frag, param_frag[1][compute_idx], acc_frag);
        }
        wmma::store_matrix_sync(out + begin + IDX(HALO + col / 7, HALO, ldm), acc_frag, TENSOR_CORE_M, wmma::mem_row_major);
    }
}


/**
 * @param in input array pointer
 * @param out output array pointer
 * @param params parameter array pointer (length 49)
 * 
*/
void gpu_box_2d1r(const real_t * __restrict__ in, real_t * __restrict__ out, const real_t * __restrict__ params, const int times, const int input_m, const int input_n) {
    real_t param_matrix_h[2][MMA_NUM * TENSOR_CORE_M * TENSOR_CORE_K] = {0.0};

    // Build Weight Matrix A
    for (int col = 0; col < TENSOR_CORE_M; col++) {
        for(int i = 0; i < UNIT_LENGTH; i++) {
            for(int j = 0; j < UNIT_LENGTH; j++) {
                if (j >= col) {
                    int idx = (i * UNIT_LENGTH + j) * TENSOR_CORE_M + col;
                    param_matrix_h[0][idx] = params[i * UNIT_LENGTH + j - col];
                    param_matrix_h[0][idx+8] = params[i * UNIT_LENGTH + j - col];
                }
            }
        }
    }
    // Build Weight Matrix B
    for (int col = 0; col < TENSOR_CORE_M; col++) {
        for(int i = 0; i < UNIT_LENGTH; i++) {
            for(int j = 0; j < UNIT_LENGTH; j++) {
                if (j < col - 8) {
                    int idx = (i * UNIT_LENGTH + j) * TENSOR_CORE_M + col;
                    param_matrix_h[1][idx] = params[i * UNIT_LENGTH + j - col + 15];
                    param_matrix_h[1][idx-8] = params[i * UNIT_LENGTH + j - col + 15];
                }
            }
        }
    }

    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(param_matrix_d), param_matrix_h, sizeof(param_matrix_h)));

    #ifdef DEBUG

    std::cout << "[Stencil Kernel]" << std::endl;
    for(int i = 0; i < 7; i++){
        for(int j = 0; j < 7; j++){
            std::cout << params[i * 7 + j] << " ";
        }
        std::cout << std::endl;
    }

    std::cout << "\n[Weight Matrix A]" << std::endl;
    for (int i = 0; i < MMA_NUM; i++) {
        int mma_offset = i* TENSOR_CORE_M * TENSOR_CORE_K;
        for(int j=0; j < TENSOR_CORE_K; j++){
            for(int k=0; k < TENSOR_CORE_M; k++){
                std::cout << param_matrix_h[0][mma_offset + j * TENSOR_CORE_M + k] << " ";
            }
            std::cout << std::endl;
        }
    }

    std::cout << "\n[Weight Matrix B]" << std::endl;
    for (int i = 0; i < MMA_NUM; i++) {
        int mma_offset = i * TENSOR_CORE_M * TENSOR_CORE_K;
        for(int j = 0; j < TENSOR_CORE_K; j++){
            for(int k = 0; k < TENSOR_CORE_M; k++){
                std::cout << param_matrix_h[1][mma_offset + j * TENSOR_CORE_M + k] << " ";
            }
            std::cout << std::endl;
        }
    }
    std::cout << std::endl;
    
    #endif

    const int rows = input_m + 2 * HALO;
    const int cols = input_n + 2 * HALO + 2;
    const size_t array_size = rows * cols * sizeof(real_t);
    real_t *  array_d[2];
    CUDA_CHECK(hipMalloc(&array_d[0], array_size));
    CUDA_CHECK(hipMalloc(&array_d[1], array_size));
    CUDA_CHECK(hipMemset(array_d[0], 0, array_size));
    CUDA_CHECK(hipMemcpy(array_d[0], in, array_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(array_d[1], 0, array_size));

    
    const int BLOCK_M = (input_m + BLOCK_SIZE_ROW - 1) / BLOCK_SIZE_ROW; 
    const int BLOCK_N = (input_n + BLOCK_SIZE_COL - 1) / BLOCK_SIZE_COL; 
    dim3 grid_config(BLOCK_M, BLOCK_N);
    dim3 block_config(32 * WARP_PER_BLOCK);

    // Lookup tables (with linearized indices)
    int lookup_table1_h[D_BLOCK_SIZE_ROW][D_BLOCK_SIZE_COL];
    int lookup_table2_h[D_BLOCK_SIZE_ROW][D_BLOCK_SIZE_COL];
    for (int i = 0; i < D_BLOCK_SIZE_ROW; i++) {
        for (int j = 0; j < D_BLOCK_SIZE_COL; j++) {
            // Stencil2row Matrix A
            if ((j + 1) % 8 != 0 && j < D_BLOCK_SIZE_COL - 2 * HALO - 1) {
                lookup_table1_h[i][j] = IDX(j / (UNIT_LENGTH + 1), UNIT_LENGTH * i + j % (UNIT_LENGTH + 1), SM_SIZE_COL);
            } else {
                lookup_table1_h[i][j] = SM_SIZE_ROW * SM_SIZE_COL - 1;
            }
            // Stencil2row Matrix B
            if ((j + 2) % 8 != 0 && j > 2 * HALO) {
                lookup_table2_h[i][j] = IDX((j - UNIT_LENGTH) / (UNIT_LENGTH + 1), UNIT_LENGTH * i + (j - UNIT_LENGTH) % (UNIT_LENGTH + 1), SM_SIZE_COL);
            } else {
                lookup_table2_h[i][j] = SM_SIZE_ROW * SM_SIZE_COL - 1;
            }
        }
    }

    #ifdef DEBUG

    std::cout << "\nSharedmem[0]" << std::endl;
    float debug_sharedmem[2][SM_SIZE_ROW * SM_SIZE_COL] = {0.0};

    for (int i = 0; i < D_BLOCK_SIZE_ROW; i++)
    {
        for(int j = 0; j < D_BLOCK_SIZE_COL; j++) 
        {
            debug_sharedmem[0][lookup_table1_h[i][j]] = in[IDX(i, j, cols)];
            debug_sharedmem[1][lookup_table2_h[i][j]] = in[IDX(i, j, cols)];
        }
    }

    for (int i = 0; i < SM_SIZE_ROW; i++)
    {
        for(int j = 0; j < SM_SIZE_COL; j++) 
        {
            std::cout << debug_sharedmem[0][i * SM_SIZE_COL + j] << " ";
            //sharedmem[0][lookup_table1[i]] = in[begin + IDX(row, col, ldm)];
        }
        std::cout << std::endl;
    }

    for (int i = 0; i < D_BLOCK_SIZE_ROW; i++)
    {
        for(int j = 0; j < D_BLOCK_SIZE_COL; j++) 
        {
            std::cout << in[IDX(i, j, cols)] << " ";
            //sharedmem[1][lookup_table2[i]] = in[begin + IDX(row, col, ldm)];
        }
        std::cout << std::endl;
    }

    #endif

    int * lookup_table1_d;
    int * lookup_table2_d;
    CUDA_CHECK(hipMalloc(&lookup_table1_d, D_BLOCK_SIZE_ROW * D_BLOCK_SIZE_COL * sizeof(int)));
    CUDA_CHECK(hipMalloc(&lookup_table2_d, D_BLOCK_SIZE_ROW * D_BLOCK_SIZE_COL * sizeof(int)));
    CUDA_CHECK(hipMemcpy(lookup_table1_d, lookup_table1_h, D_BLOCK_SIZE_ROW * D_BLOCK_SIZE_COL * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(lookup_table2_d, lookup_table2_h, D_BLOCK_SIZE_ROW * D_BLOCK_SIZE_COL * sizeof(int), hipMemcpyHostToDevice));
    

    // timing
    int i = 0;

    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

    for(; i < times; i++) {
        #ifdef USE_DOUBLE_PRECISION
            CUDAKERNELCHECK((kernel2d_fp64<<<grid_config, block_config>>>(array_d[i % 2], array_d[(i + 1) % 2], cols, lookup_table1_d, lookup_table2_d)));
        #endif
        #ifdef USE_FLOAT_PRECISION
            CUDAKERNELCHECK((kernel2d_fp32<<<grid_config, block_config>>>(array_d[i % 2], array_d[(i + 1) % 2], cols, lookup_table1_d, lookup_table2_d)));
        #endif
        }
    CUDA_CHECK(hipDeviceSynchronize());

    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    std::cout << "ConvStencil(2D): " << std::endl;
    std::cout << "Time = " << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() << "[ms]" << std::endl;
    
    double secs = std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() / 1e6;

    printf("GStencil/s = %f\n", ((double)input_m * input_n * times * 3) / secs / 1e9);
    
    std::ofstream csv("logs/logs.csv", std::ios::app);
    csv << "ConvStencil(2D),star_2d1r," << input_m << "," << times << "," << precision_name(out[0]) << ","
        << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << "," 
        << ((double)input_m * input_n * times * 3) / secs / 1e9 << std::endl;


    CUDA_CHECK(hipMemcpy(out, array_d[i % 2], array_size, hipMemcpyDeviceToHost));

    return;
}


/*
__global__ void kernel2d_fp64 (const double * __restrict__ in, double * __restrict__ out, const int ldm, const int * __restrict__ lookup_table1, const int * __restrict__ lookup_table2) {
    __shared__ double sharedmem[2][SM_SIZE_ROW * SM_SIZE_COL];
    int begin = IDX(blockIdx.x * BLOCK_SIZE_ROW, blockIdx.y * BLOCK_SIZE_COL + 1, ldm);
    int tid = threadIdx.x;
    int totalThreads = blockDim.x;
#pragma unroll
    for (int i = tid; i < D_BLOCK_SIZE_ROW * D_BLOCK_SIZE_COL; i += totalThreads) {
        int row = i / D_BLOCK_SIZE_COL;
        int col = i % D_BLOCK_SIZE_COL;
        sharedmem[0][lookup_table1[i]] = in[begin + IDX(row, col, ldm)];
        sharedmem[1][lookup_table2[i]] = in[begin + IDX(row, col, ldm)];
    }
    __syncthreads();


    int warp_id = threadIdx.x / 32;

    nvcuda::wmma::fragment<wmma::matrix_b, 8, 8, 4, double, wmma::row_major> param_frag[2][MMA_NUM];
#pragma unroll
    for (int i = 0; i < MMA_NUM; i++) {
        nvcuda::wmma::load_matrix_sync(param_frag[0][i], param_matrix_d + i * 32, 8);
        nvcuda::wmma::load_matrix_sync(param_frag[1][i], param_matrix_d + 52 * 8 + i * 32, 8);
    }

    wmma::fragment<wmma::accumulator, 8, 8, 4, double> acc_frag;
    wmma::fragment<wmma::matrix_a, 8, 8, 4, double, wmma::row_major> in_frag;
    for (int col = warp_id * 28; col < warp_id * 28 + 28; col += UNIT_LENGTH) {
        wmma::fill_fragment(acc_frag, 0.0);
#pragma unroll
        for (int compute_idx = 0; compute_idx < MMA_NUM; compute_idx++) {
            wmma::load_matrix_sync(in_frag, sharedmem[0] + IDX(0, col + compute_idx * 4, SM_SIZE_COL), SM_SIZE_COL);
            wmma::mma_sync(acc_frag, in_frag, param_frag[0][compute_idx], acc_frag);
        }
#pragma unroll
        for (int compute_idx = 0; compute_idx < MMA_NUM; compute_idx++) {
            wmma::load_matrix_sync(in_frag, sharedmem[1] + IDX(0, col + compute_idx * 4, SM_SIZE_COL), SM_SIZE_COL);
            wmma::mma_sync(acc_frag, in_frag, param_frag[1][compute_idx], acc_frag);
        }
        wmma::store_matrix_sync(out + begin + IDX(HALO + col / 7, HALO, ldm), acc_frag, TENSOR_CORE_M, wmma::mem_row_major);
    }
}
*/